#include "hip/hip_runtime.h"
// CIS565 CUDA Rasterizer: A simple rasterization pipeline for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/random.h>
#include "rasterizeKernels.h"
#include "rasterizeTools.h"

glm::vec3 *framebuffer;
fragment *depthbuffer;
float *device_vbo;
float *device_cbo;
int *device_ibo;
triangle* primitives;
float *device_vbo_window_coords;

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//Handy dandy little hashing function that provides seeds for random number generation
__host__ __device__ unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Writes a given fragment to a fragment buffer at a given location
__host__ __device__ void writeToDepthbuffer(int x, int y, fragment frag, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    depthbuffer[index] = frag;
  }
}

//Reads a fragment from a given location in a fragment buffer
__host__ __device__ fragment getFromDepthbuffer(int x, int y, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return depthbuffer[index];
  }else{
    fragment f;
    return f;
  }
}

//Writes a given pixel to a pixel buffer at a given location
__host__ __device__ void writeToFramebuffer(int x, int y, glm::vec3 value, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    framebuffer[index] = value;
  }
}

//Reads a pixel from a pixel buffer at a given location
__host__ __device__ glm::vec3 getFromFramebuffer(int x, int y, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return framebuffer[index];
  }else{
    return glm::vec3(0,0,0);
  }
}

//Kernel that clears a given pixel buffer with a given color
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image, glm::vec3 color){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = color;
    }
}

//Kernel that clears a given fragment buffer with a given fragment
__global__ void clearDepthBuffer(glm::vec2 resolution, fragment* buffer, fragment frag){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      fragment f = frag;
      f.position.x = x;
      f.position.y = y;
      buffer[index] = f;
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;      
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}


/*********** DANNY'S PRIMARY CONTRIBUTION - START ***********/

// TODO: Implement a vertex shader.
// Convert vertices from model-space to clip-space.
__global__
void vertexShadeKernel( float *vbo,
						int vbosize,
						glm::mat4 mvp_matrix,
						glm::vec2 resolution,
						float *vbo_window_coords )
{
	int index = ( blockIdx.x * blockDim.x ) + threadIdx.x;

	// Divide by 3 because each vertex has 3 components (x, y, and z).
	if ( index < vbosize / 3 ) {
		// Create point to transform.
		int vbo_index = index * 3;
		glm::vec4 v( vbo[vbo_index + 0], vbo[vbo_index + 1], vbo[vbo_index + 2], 1.0f );

		// Transform point from object-space to clip-space by multiplying by the composite model, view, projection matrices.
		glm::vec4 vt = mvp_matrix * v;

		// Transform point to NDC-space by dividing x-, y-, and z-components by w-component (perspective division).
		// [-1, 1].
		glm::vec3 v_ndc( vt.x / vt.w, vt.y / vt.w, vt.z / vt.w );

		// Transform x and y range from [-1, 1] to [0, 1].
		glm::vec2 v_remapped( ( v_ndc.x + 1.0f ) / 2.0f, ( v_ndc.y + 1.0f ) / 2.0f );

		// Transform x- and y-coordinates to window-space.
		glm::vec2 v_window( v_remapped.x * resolution.x, v_remapped.y * resolution.y );

		// Save transformed vertices.
		vbo_window_coords[vbo_index + 0] = v_window.x;
		vbo_window_coords[vbo_index + 1] = v_window.y;
		vbo_window_coords[vbo_index + 2] = v_ndc.z;
	}
}


// TODO: Implement primative assembly.
__global__
void primitiveAssemblyKernel( float *vbo,
							  int vbosize,
							  float *cbo,
							  int cbosize,
							  int *ibo,
							  int ibosize,
							  triangle *primitives )
{

	// TODO: Convert vertices to primitives (triangles or fragments).

	int index = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	int primitivesCount = ibosize / 3;
	if ( index < primitivesCount ) {

	}
}


// TODO: Implement a rasterization method, such as scanline.
__global__
void rasterizationKernel( triangle *primitives,
						  int primitivesCount,
						  fragment *depthbuffer,
						  glm::vec2 resolution )
{

	// TODO: Which pixels does a primitive cover?

	int index = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	if ( index < primitivesCount ) {

	}
}


// TODO: Implement a fragment shader.
__global__
void fragmentShadeKernel( fragment *depthbuffer,
						  glm::vec2 resolution )
{

	// TODO: How does light interact with a fragment?
	// TODO: Write pixel color to frame buffer.

	int x = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	int y = ( blockIdx.y * blockDim.y ) + threadIdx.y;
	int index = x + ( y * resolution.x );
	if ( x <= resolution.x && y <= resolution.y ) {

	}
}

/*********** DANNY'S PRIMARY CONTRIBUTION - END ***********/


//Writes fragment colors to the framebuffer
__global__ void render(glm::vec2 resolution, fragment* depthbuffer, glm::vec3* framebuffer){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if(x<=resolution.x && y<=resolution.y){
    framebuffer[index] = depthbuffer[index].color;
  }
}

// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRasterizeCore( uchar4 *PBOpos,
						float frame,
						float *vbo,
						int vbosize,
						float *cbo,
						int cbosize,
						int *ibo,
						int ibosize,
						simpleCamera camera )
{
	// set up crucial magic
	int tileSize = 8;
	dim3 threadsPerBlock( tileSize,
						  tileSize );
	dim3 fullBlocksPerGrid( ( int )ceil( ( float )camera.resolution.x / ( float )tileSize ),
							( int )ceil( ( float )camera.resolution.y / ( float )tileSize ) );

	// set up framebuffer
	framebuffer = NULL;
	hipMalloc( ( void** )&framebuffer,
				( int )camera.resolution.x * ( int )camera.resolution.y * sizeof( glm::vec3 ) );
  
	// set up depthbuffer
	depthbuffer = NULL;
	hipMalloc( ( void** )&depthbuffer,
				( int )camera.resolution.x * ( int )camera.resolution.y * sizeof( fragment ) );

	// kernel launches to black out accumulated/unaccumlated pixel buffers and clear our scattering states
	clearImage<<< fullBlocksPerGrid, threadsPerBlock >>>( camera.resolution,
														  framebuffer,
														  glm::vec3( 0.0f, 0.0f, 0.0f ) );
  
	fragment frag;
	frag.color = glm::vec3( 0.0f, 0.0f, 0.0f );
	frag.normal = glm::vec3( 0.0f, 0.0f, 0.0f );
	frag.position = glm::vec3( 0.0f, 0.0f, -10000.0f );
	clearDepthBuffer<<< fullBlocksPerGrid, threadsPerBlock >>>( camera.resolution,
																depthbuffer,
																frag );

	//------------------------------
	// memory stuff
	//------------------------------
	primitives = NULL;
	hipMalloc( ( void** )&primitives,
				( ibosize / 3 ) * sizeof( triangle ) );

	device_ibo = NULL;
	hipMalloc( ( void** )&device_ibo,
				ibosize * sizeof( int ) );
	hipMemcpy( device_ibo,
				ibo,
				ibosize * sizeof( int ),
				hipMemcpyHostToDevice );

	device_vbo = NULL;
	hipMalloc( ( void** )&device_vbo,
				vbosize * sizeof( float ) );
	hipMemcpy( device_vbo,
				vbo,
				vbosize * sizeof( float ),
				hipMemcpyHostToDevice );

	device_vbo_window_coords = NULL;
	hipMalloc( ( void** )&device_vbo_window_coords,
				vbosize * sizeof( float ) );

	device_cbo = NULL;
	hipMalloc( ( void** )&device_cbo,
				cbosize * sizeof( float ) );
	hipMemcpy( device_cbo,
				cbo,
				cbosize * sizeof( float ),
				hipMemcpyHostToDevice );

	tileSize = 32;
	int primitiveBlocks = ceil( ( ( float )vbosize / 3 ) / ( ( float )tileSize ) );

	//------------------------------
	// vertex shader
	//------------------------------

	// Define model matrix.
	// Transforms from object-space to world-space.
	glm::mat4 model_matrix( 1.0f ); // Identity matrix.
	
	// Define view matrix.
	// Transforms from world-space to camera-space.
	glm::mat4 view_matrix = glm::lookAt( camera.position,
										 camera.target,
										 camera.up );

	// Define projection matrix.
	// Transforms from camera-space to clip-space.
	glm::mat4 projection_matrix = glm::perspective( camera.fov_y,
													camera.resolution.x / camera.resolution.y,
													camera.near_clip,
													camera.far_clip );

	vertexShadeKernel<<< primitiveBlocks, tileSize >>>( device_vbo,
														vbosize,
														projection_matrix * view_matrix * model_matrix,
														camera.resolution,
														device_vbo_window_coords );
	hipDeviceSynchronize();

	//------------------------------
	// primitive assembly
	//------------------------------
	primitiveBlocks = ceil( ( ( float )ibosize / 3 ) / ( ( float )tileSize ) );
	primitiveAssemblyKernel<<< primitiveBlocks, tileSize >>>( device_vbo,
															  vbosize,
															  device_cbo,
															  cbosize,
															  device_ibo,
															  ibosize,
															  primitives );
	hipDeviceSynchronize();

	//------------------------------
	// rasterization
	//------------------------------
	rasterizationKernel<<< primitiveBlocks, tileSize >>>( primitives,
														  ibosize / 3,
														  depthbuffer,
														  camera.resolution );
	hipDeviceSynchronize();

	//------------------------------
	// fragment shader
	//------------------------------
	fragmentShadeKernel<<< fullBlocksPerGrid, threadsPerBlock >>>( depthbuffer,
																   camera.resolution );
	hipDeviceSynchronize();

	//------------------------------
	// write fragments to framebuffer
	//------------------------------
	render<<< fullBlocksPerGrid, threadsPerBlock >>>( camera.resolution,
													  depthbuffer,
													  framebuffer );
	sendImageToPBO<<< fullBlocksPerGrid, threadsPerBlock >>>( PBOpos,
															  camera.resolution,
															  framebuffer );
	hipDeviceSynchronize();

	kernelCleanup();
	checkCUDAError("Kernel failed!");
}

void kernelCleanup(){
  hipFree( primitives );
  hipFree( device_vbo );
  hipFree( device_cbo );
  hipFree( device_ibo );
  hipFree( framebuffer );
  hipFree( depthbuffer );
  hipFree( device_vbo_window_coords );
}