#include "hip/hip_runtime.h"
// CIS565 CUDA Rasterizer: A simple rasterization pipeline for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/count.h>
#include "rasterizeKernels.h"
#include "rasterizeTools.h"

glm::vec3* framebuffer;
fragment* depthbuffer;
float* device_vbo;
float* device_nbo;
float* device_cbo;
int* device_ibo;
triangle* primitives;

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

__host__ __device__ void screenToNDC(int x, int resolution, float* ndcX) {
  *ndcX = - 2 * (x / (float)resolution - 0.5f);
}

__host__ __device__ void ndcToScreen(float ndcX, int resolution, int* x) {
  *x = -(ndcX - 1) * resolution / 2;
}

//Handy dandy little hashing function that provides seeds for random number generation
__host__ __device__ unsigned int hash(unsigned int a){
  a = (a+0x7ed55d16) + (a<<12);
  a = (a^0xc761c23c) ^ (a>>19);
  a = (a+0x165667b1) + (a<<5);
  a = (a+0xd3a2646c) ^ (a<<9);
  a = (a+0xfd7046c5) + (a<<3);
  a = (a^0xb55a4f09) ^ (a>>16);
  return a;
}

//Writes a given fragment to a fragment buffer at a given location
__host__ __device__ void writeToDepthbuffer(int x, int y, fragment frag, fragment* depthbuffer, glm::vec2 resolution){
  if(x>0 && x<resolution.x && y>0 && y<resolution.y){
    int index = (y*resolution.x) + x;
    depthbuffer[index] = frag;
  }
}

//Reads a fragment from a given location in a fragment buffer
__host__ __device__ fragment getFromDepthbuffer(int x, int y, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return depthbuffer[index];
  }else{
    fragment f;
    return f;
  }
}

//Writes a given pixel to a pixel buffer at a given location
__host__ __device__ void writeToFramebuffer(int x, int y, glm::vec3 value, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    framebuffer[index] = value;
  }
}

//Reads a pixel from a pixel buffer at a given location
__host__ __device__ glm::vec3 getFromFramebuffer(int x, int y, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return framebuffer[index];
  }else{
    return glm::vec3(0,0,0);
  }
}

//Kernel that clears a given pixel buffer with a given color
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image, glm::vec3 color){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  if(x<=resolution.x && y<=resolution.y){
    image[index] = color;
  }
}

//Kernel that clears a given fragment buffer with a given fragment
__global__ void clearDepthBuffer(glm::vec2 resolution, fragment* buffer, fragment frag){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  if(x<=resolution.x && y<=resolution.y){
    fragment f = frag;
    f.position.x = x;
    f.position.y = y;
    buffer[index] = f;
  }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if(x<=resolution.x && y<=resolution.y){

    glm::vec3 color;      
    color.x = image[index].x*255.0;
    color.y = image[index].y*255.0;
    color.z = image[index].z*255.0;

    if(color.x>255){
      color.x = 255;
    }

    if(color.y>255){
      color.y = 255;
    }

    if(color.z>255){
      color.z = 255;
    }

    // Each thread writes one pixel location in the texture (textel)
    PBOpos[index].w = 0;
    PBOpos[index].x = color.x;     
    PBOpos[index].y = color.y;
    PBOpos[index].z = color.z;
  }
}

//TODO: Implement a vertex shader
__global__ void vertexShadeKernel(float* vbo, int vbosize, glm::mat4 mvp){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index<vbosize/3){
    glm::vec4 p (vbo[index * 3], vbo[index * 3 + 1], vbo[index * 3 + 2], 1);
    p = mvp * p;
    vbo[index * 3] = p.x / p.w;
    vbo[index * 3 + 1] = p.y / p.w;
    vbo[index * 3 + 2] = p.z / p.w;
  }
}

//TODO: Implement primative assembly
__global__ void primitiveAssemblyKernel(float* vbo, int vbosize, float* nbo, int nbosize, float* cbo, int cbosize, int* ibo, int ibosize, triangle* primitives){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int primitivesCount = ibosize/3;
  if(index<primitivesCount){
    triangle tri;
    int i = ibo[index * 3] * 3;
    tri.p0 = glm::vec3(vbo[i], vbo[i + 1], vbo[i + 2]);
    tri.n0 = glm::vec3(nbo[i], nbo[i + 1], nbo[i + 2]);
    tri.c0 = glm::vec3(cbo[i], cbo[i + 1], cbo[i + 2]);
    i = ibo[index * 3 + 1] * 3;
    tri.p1 = glm::vec3(vbo[i], vbo[i + 1], vbo[i + 2]);
    tri.n1 = glm::vec3(nbo[i], nbo[i + 1], nbo[i + 2]);
    tri.c1 = glm::vec3(cbo[i], cbo[i + 1], cbo[i + 2]);
    i = ibo[index * 3 + 2] * 3;
    tri.p2 = glm::vec3(vbo[i], vbo[i + 1], vbo[i + 2]);
    tri.n2 = glm::vec3(nbo[i], nbo[i + 1], nbo[i + 2]);
    tri.c2 = glm::vec3(cbo[i], cbo[i + 1], cbo[i + 2]);
    /*glm::vec3 normal = glm::normalize(glm::cross(tri.p1 - tri.p0, tri.p2 - tri.p0) +
                                      glm::cross(tri.p2 - tri.p1, tri.p0 - tri.p1) +
                                      glm::cross(tri.p0 - tri.p2, tri.p1 - tri.p2));
    tri.n0 = tri.n1 = tri.n2 = normal;*/
    primitives[index] = tri;
  }
}

__global__ void backfaceCullingKernel(triangle* primitives, int primitivesCount) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index<primitivesCount){

  }
}

//TODO: Implement a rasterization method, such as scanline.
__global__ void rasterizationKernel(triangle* primitives, int primitivesCount, fragment* depthbuffer, glm::vec2 resolution){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index<primitivesCount){
    // Draw Vertices
    /*
    int x, y;
    ndcToScreen(primitives[index].p0.x, resolution.x, &x);
    ndcToScreen(primitives[index].p0.y, resolution.y, &y);
            
    fragment frag;
    frag.color = primitives[index].c0;
    frag.normal = primitives[index].n0;
    frag.position = primitives[index].p0;
    writeToDepthbuffer(x, y, frag, depthbuffer, resolution);
    
    ndcToScreen(primitives[index].p1.x, resolution.x, &x);
    ndcToScreen(primitives[index].p1.y, resolution.y, &y);
            
    frag.color = primitives[index].c1;
    frag.normal = primitives[index].n1;
    frag.position = primitives[index].p1;
    writeToDepthbuffer(x, y, frag, depthbuffer, resolution);
    
    ndcToScreen(primitives[index].p2.x, resolution.x, &x);
    ndcToScreen(primitives[index].p2.y, resolution.y, &y);
            
    frag.color = primitives[index].c2;
    frag.normal = primitives[index].n2;
    frag.position = primitives[index].p2;
    writeToDepthbuffer(x, y, frag, depthbuffer, resolution);
    */

    // Draw Faces
    triangle t = primitives[index];
    point top, middle, bottom;
    top.position = t.p0;
    middle.position = t.p1;
    bottom.position = t.p2;
    top.color = t.c0;
    middle.color = t.c1;
    bottom.color = t.c2;
    top.normal = t.n0;
    middle.normal = t.n1;
    bottom.normal = t.n2;

    point temp;

    // Do a basic bubble sort
    for (int i = 0; i < 2; i++) {
      if (bottom.position.y > middle.position.y) {
        temp = bottom;
        bottom = middle;
        middle = temp;
      }
      if (middle.position.y > top.position.y) {
        temp = middle;
        middle = top;
        top = temp;
      }
    }

    // Ignore triangle if it's outside
    // TODO: move this to a clipper later, with the x and z clipping as well.
    if (top.position.y < -1 || bottom.position.y > 1) {
      return;
    }

    // "left" and "right" are relative to each other, not top.
    point pointLeft, pointRight;    // used for interpolation

    if (bottom.position.x > middle.position.x) {  // top->middle is on the left
      pointLeft = middle;
      pointRight = bottom;
    } else {        // top->bottom is on left
      pointLeft = bottom;
      pointRight = middle;
    }
    
    float currNDCx = top.position.x;
    float currNDCy = top.position.y;
    int currY, currX;
    ndcToScreen(currNDCx, resolution.x, &currX);
    ndcToScreen(currNDCy, resolution.y, &currY);

    while (currNDCy > middle.position.y && currNDCy > -1) {
      // only perform these operations if the current y coordinate is in the screen.
      if (currNDCy <= 1) {
        // interpolate along the edges
        float tLeft = (top.position.y - currNDCy) / (top.position.y - pointLeft.position.y);
        if (top.position.y == pointLeft.position.y) {
          tLeft = 0;
        }
        float tRight = (top.position.y - currNDCy) / (top.position.y - pointRight.position.y);
        if (top.position.y == pointRight.position.y) {
          tRight = 0;
        }

        // Would saving 1-tleft and 1-tright into variables save 1 cycle per statement (if they fit in registers?)
        glm::vec3 cLeft, cRight, pLeft, pRight, nLeft, nRight;
        pLeft = (1 - tLeft) * top.position + tLeft * pointLeft.position;
        nLeft = (1 - tLeft) * top.normal + tLeft * pointLeft.normal;
        cLeft = (1 - tLeft) * top.color + tLeft * pointLeft.color;
        pRight = (1 - tRight) * top.position + tRight * pointRight.position;
        nRight = (1 - tRight) * top.normal + tRight * pointRight.normal;
        cRight = (1 - tRight) * top.color + tRight * pointRight.color;
        int rBound = 0;
        int lBound = 0;
        ndcToScreen(pRight.x, resolution.x, &rBound);
        ndcToScreen(pLeft.x, resolution.x, &lBound);
        for (currX = lBound; currX >= rBound; currX--) {
          if (currX >= 0 && currX < resolution.x) {
            screenToNDC(currX, resolution.x, &currNDCx);
            // interpolate color, normal, and position
            float t = (currX - lBound) / (rBound - lBound);
            if (pRight.x == pLeft.x) {
              t = 0;
            }
            fragment frag;
            frag.position = (1 - t) * pLeft + t * pRight;
            frag.normal = (1 - t) * nLeft + t * nRight;
            frag.color = (1 - t) * cLeft + t * cRight;
            writeToDepthbuffer(currX, currY, frag, depthbuffer, resolution);
          }
        }
      }
      currY++;
      screenToNDC(currY, resolution.y, &currNDCy);
    }
    
    if (middle.position.x < top.position.x) {
      pointLeft = middle;
      pointRight = top;
    } else {
      pointLeft = top;
      pointRight = middle;
    }

    while (currNDCy > bottom.position.y && currNDCy > -1) {
      // only perform these operations if the current y coordinate is in the screen.
      if (currNDCy <= 1) {
        // interpolate along the edges
        float tLeft = (pointLeft.position.y - currNDCy) / (pointLeft.position.y - bottom.position.y);
        if (pointLeft.position.y == bottom.position.y) {
          tLeft = 0;
        }
        float tRight = (pointRight.position.y - currNDCy) / (pointRight.position.y - bottom.position.y);
        if (pointRight.position.y == bottom.position.y) {
          tRight = 0;
        }

        // Would saving 1-tleft and 1-tright into variables save 1 cycle per statement (if they fit in registers?)
        glm::vec3 cLeft, cRight, pLeft, pRight, nLeft, nRight;
        pLeft = (1 - tLeft) * pointLeft.position + tLeft * bottom.position;
        nLeft = (1 - tLeft) * pointLeft.normal + tLeft * bottom.normal;
        cLeft = (1 - tLeft) * pointLeft.color + tLeft * bottom.color;
        pRight = (1 - tRight) * pointRight.position + tRight * bottom.position;
        nRight = (1 - tRight) * pointRight.normal + tRight * bottom.normal;
        cRight = (1 - tRight) * pointRight.color + tRight * bottom.color;

        int rBound = 0;
        int lBound = 0;
        ndcToScreen(pRight.x, resolution.x, &rBound);
        ndcToScreen(pLeft.x, resolution.x, &lBound);
        for (currX = lBound; currX >= rBound; currX--) {
          if (currX >= 0 && currX < resolution.x) {
            screenToNDC(currX, resolution.x, &currNDCx);
            // interpolate color, normal, and position
            float t = (currX - lBound) / (rBound - lBound);
            if (pRight.x == pLeft.x) {
              t = 0;
            }
            fragment frag;
            frag.position = (1 - t) * pLeft + t * pRight;
            frag.normal = (1 - t) * nLeft + t * nRight;
            frag.color = (1 - t) * cLeft + t * cRight;
            writeToDepthbuffer(currX, currY, frag, depthbuffer, resolution);
          }
        }
      }
      currY++;
      screenToNDC(currY, resolution.y, &currNDCy);
    }
  }
}

//TODO: Implement a fragment shader
// Modifies the .color value per fragment.
// Simple Blinn-Phong shading, light needs to be transformed into clip coordinates.
__global__ void fragmentShadeKernel(fragment* depthbuffer, glm::vec2 resolution, light light, glm::mat4 matVPinv){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  if(x<=resolution.x && y<=resolution.y){
    fragment f = depthbuffer[index];
    if (f.position.z > 0) { //ignore all the empty space (z = -10000)
      glm::vec4 origPos = matVPinv * glm::vec4(f.position,1);
      float diffuse = glm::dot(f.normal, glm::normalize(light.position - glm::vec3(origPos / origPos.w)));
      if (diffuse < 0) {
        diffuse = 0;
      }
      depthbuffer[index].color *= light.color * diffuse;
    }
  }
}

//Writes fragment colors to the framebuffer
__global__ void render(glm::vec2 resolution, fragment* depthbuffer, glm::vec3* framebuffer){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if(x<=resolution.x && y<=resolution.y){
    // Color
    framebuffer[index] = depthbuffer[index].color;
    // Normal
    //framebuffer[index] = depthbuffer[index].normal;
    //framebuffer[index] = glm::normalize(glm::vec3(depthbuffer[index].normal.r, depthbuffer[index].normal.g, 0));
    // Distance
    //framebuffer[index] = glm::vec3(depthbuffer[index].position.z);
  }
}

struct clippingOrBackface {
  __host__ __device__
    bool operator() (const triangle t) {
      glm::vec3 normal = glm::normalize(glm::cross(t.p1 - t.p0, t.p2 - t.p0) +
                                      glm::cross(t.p2 - t.p1, t.p0 - t.p1) +
                                      glm::cross(t.p0 - t.p2, t.p1 - t.p2));
      return (normal.z < 0) || ((t.p0.x > 1 || t.p0.x < -1) && (t.p0.y > 1 || t.p0.y < -1) &&
                              (t.p1.x > 1 || t.p1.x < -1) && (t.p1.y > 1 || t.p1.y < -1) &&
                              (t.p2.x > 1 || t.p2.x < -1) && (t.p2.y > 1 || t.p2.y < -1));
  }
};

// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRasterizeCore(uchar4* PBOpos, glm::vec2 resolution, float frame, float* vbo, int vbosize, float* nbo, int nbosize, float* cbo, int cbosize, int* ibo, int ibosize){

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(resolution.x)/float(tileSize)), (int)ceil(float(resolution.y)/float(tileSize)));

  //set up framebuffer
  framebuffer = NULL;
  hipMalloc((void**)&framebuffer, (int)resolution.x*(int)resolution.y*sizeof(glm::vec3));

  //set up depthbuffer
  depthbuffer = NULL;
  hipMalloc((void**)&depthbuffer, (int)resolution.x*(int)resolution.y*sizeof(fragment));

  //kernel launches to black out accumulated/unaccumlated pixel buffers and clear our scattering states
  clearImage<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, framebuffer, glm::vec3(0,0,0));

  fragment frag;
  frag.color = glm::vec3(0,0,0);
  frag.normal = glm::vec3(0,0,0);
  frag.position = glm::vec3(0,0,-10000);
  clearDepthBuffer<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer,frag);

  //------------------------------
  //memory stuff
  //------------------------------
  primitives = NULL;
  hipMalloc((void**)&primitives, (ibosize/3)*sizeof(triangle));

  device_ibo = NULL;
  hipMalloc((void**)&device_ibo, ibosize*sizeof(int));
  hipMemcpy( device_ibo, ibo, ibosize*sizeof(int), hipMemcpyHostToDevice);

  device_vbo = NULL;
  hipMalloc((void**)&device_vbo, vbosize*sizeof(float));
  hipMemcpy( device_vbo, vbo, vbosize*sizeof(float), hipMemcpyHostToDevice);

  device_nbo = NULL;
  hipMalloc((void**)&device_nbo, nbosize*sizeof(float));
  hipMemcpy( device_nbo, nbo, nbosize*sizeof(float), hipMemcpyHostToDevice);

  device_cbo = NULL;
  hipMalloc((void**)&device_cbo, cbosize*sizeof(float));
  hipMemcpy( device_cbo, cbo, cbosize*sizeof(float), hipMemcpyHostToDevice);

  tileSize = 32;
  int primitiveBlocks = ceil(((float)vbosize/3)/((float)tileSize));

  //------------------------------
  //camera setup
  //------------------------------
  glm::vec3 eye (0, 0, 3);
  glm::vec3 center (0, 0, 0);
  glm::vec3 up (0, 1, 0);
  glm::mat4 matView = glm::lookAt(eye, center, up);
  float fovy, aspect, znear, zfar;
  fovy = 45;
  aspect = 1.0;
  znear = .01;
  zfar = 5;
  glm::mat4 matProj = glm::perspective(fovy, aspect, znear, zfar);

  glm::mat4 matVP = matProj * matView;

  //----------------------------
  //light setup
  //----------------------------
  light light;
  light.color = glm::vec3(1, 1, 1);
  light.position = glm::vec3(5, 0, 0);

  //------------------------------
  //vertex shader
  //------------------------------
  vertexShadeKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize, matVP);

  hipDeviceSynchronize();
  //------------------------------
  //primitive assembly
  //------------------------------
  primitiveBlocks = ceil(((float)ibosize/3)/((float)tileSize));
  primitiveAssemblyKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize, device_nbo, nbosize, device_cbo, cbosize, device_ibo, ibosize, primitives);

  hipDeviceSynchronize();

  //------------------------------
  //ez backface culling and clipping
  //------------------------------
	thrust::device_ptr<triangle> primitivesStart(primitives);

	float numRemoved = thrust::count_if(primitivesStart, primitivesStart + ibosize / 3, clippingOrBackface());
	thrust::remove_if(primitivesStart, primitivesStart + ibosize / 3, clippingOrBackface());
	float numPrimitives = ibosize / 3 - numRemoved;
  primitiveBlocks = ceil(((float)numPrimitives)/((float)tileSize));

  hipDeviceSynchronize();
  //float numPrimitives = ibosize/3;
  //------------------------------
  //rasterization
  //------------------------------
  rasterizationKernel<<<primitiveBlocks, tileSize>>>(primitives, numPrimitives, depthbuffer, resolution);

  hipDeviceSynchronize();
  //------------------------------
  //fragment shader
  //------------------------------
  fragmentShadeKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(depthbuffer, resolution, light, matVP);

  hipDeviceSynchronize();
  //------------------------------
  //write fragments to framebuffer
  //------------------------------
  render<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer, framebuffer);
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, resolution, framebuffer);

  hipDeviceSynchronize();

  kernelCleanup();

  checkCUDAError("Kernel failed!");
}

void kernelCleanup(){
  hipFree( primitives );
  hipFree( device_vbo );
  hipFree( device_nbo );
  hipFree( device_cbo );
  hipFree( device_ibo );
  hipFree( framebuffer );
  hipFree( depthbuffer );
}

