#include "hip/hip_runtime.h"
// CIS565 CUDA Rasterizer: A simple rasterization pipeline for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/random.h>
#include "rasterizeKernels.h"
#include "rasterizeTools.h"

glm::vec3* framebuffer;
fragment* depthbuffer;
bool* edgeFlag;
float* device_vbo;
float* device_cbo;
int* device_ibo;
float* device_nbo;
triangle* primitives;

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//Handy dandy little hashing function that provides seeds for random number generation
__host__ __device__ unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Writes a given fragment to a fragment buffer at a given location
__host__ __device__ void writeToDepthbuffer(int x, int y, fragment frag, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    depthbuffer[index] = frag;
  }
}

//Reads a fragment from a given location in a fragment buffer
__host__ __device__ fragment getFromDepthbuffer(int x, int y, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return depthbuffer[index];
  }else{
    fragment f;
    return f;
  }
}

//Writes a given pixel to a pixel buffer at a given location
__host__ __device__ void writeToFramebuffer(int x, int y, glm::vec3 value, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    framebuffer[index] = value;
  }
}

//Reads a pixel from a pixel buffer at a given location
__host__ __device__ glm::vec3 getFromFramebuffer(int x, int y, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return framebuffer[index];
  }else{
    return glm::vec3(0,0,0);
  }
}

//Kernel that clears a given pixel buffer with a given color
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image, glm::vec3 color){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = color;
    }
}

//Kernel that clears a given fragment buffer with a given fragment
__global__ void clearDepthBuffer(glm::vec2 resolution, fragment* buffer, fragment frag){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      fragment f = frag;
      f.position.x = x;
      f.position.y = y;
      buffer[index] = f;
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;      
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//TODO: Implement a vertex shader
__global__ void vertexShadeKernel(float* vbo, int vbosize, cudaMat4 shaderMatrix, int translateX, int translateY){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index < vbosize / 3){


	  glm::vec3 afterShader = multiplyMV(shaderMatrix, glm::vec4(vbo[3*index], vbo[3*index+1], vbo[3*index+2], 1.0f));
	  vbo[3*index] = afterShader.x + translateX;
	  vbo[3*index + 1] = afterShader.y + translateY;
	  vbo[3*index + 2] = afterShader.z;


  }
}

//TODO: Implement primative assembly
__global__ void primitiveAssemblyKernel(float* vbo, int vbosize, float* cbo, int cbosize, int* ibo, int ibosize, float* nbo, int nbosize, triangle* primitives){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int primitivesCount = ibosize/3;
  if(index < primitivesCount){
	  //primitives[index].c0.x = cbo[3 * ibo[3 * index + 0] + 0];//cbo[0]
	  //primitives[index].c0.y = cbo[3 * ibo[3 * index + 0] + 1];//cbo[1]
	  //primitives[index].c0.z = cbo[3 * ibo[3 * index + 0] + 2];//cbo[2]
	  //primitives[index].c1.x = cbo[3 * ibo[3 * index + 1] + 0];//cbo[3]
	  //primitives[index].c1.y = cbo[3 * ibo[3 * index + 1] + 1];//cbo[4]
	  //primitives[index].c1.z = cbo[3 * ibo[3 * index + 1] + 2];//cbo[5]
	  //primitives[index].c2.x = cbo[3 * ibo[3 * index + 2] + 0];//cbo[6]
	  //primitives[index].c2.y = cbo[3 * ibo[3 * index + 2] + 1];//cbo[7]
	  //primitives[index].c2.z = cbo[3 * ibo[3 * index + 2] + 2];//cbo[8]

	  primitives[index].p0.x = vbo[3 * ibo[3 * index + 0] + 0];
	  primitives[index].p0.y = vbo[3 * ibo[3 * index + 0] + 1];
	  primitives[index].p0.z = vbo[3 * ibo[3 * index + 0] + 2];
	  primitives[index].p1.x = vbo[3 * ibo[3 * index + 1] + 0];
	  primitives[index].p1.y = vbo[3 * ibo[3 * index + 1] + 1];
	  primitives[index].p1.z = vbo[3 * ibo[3 * index + 1] + 2];
	  primitives[index].p2.x = vbo[3 * ibo[3 * index + 2] + 0];
	  primitives[index].p2.y = vbo[3 * ibo[3 * index + 2] + 1];
	  primitives[index].p2.z = vbo[3 * ibo[3 * index + 2] + 2];

	  primitives[index].n0.x = nbo[3 * ibo[3 * index + 0] + 0];
	  primitives[index].n0.y = nbo[3 * ibo[3 * index + 0] + 1];
	  primitives[index].n0.z = nbo[3 * ibo[3 * index + 0] + 2];
	  primitives[index].n1.x = nbo[3 * ibo[3 * index + 1] + 0];
	  primitives[index].n1.y = nbo[3 * ibo[3 * index + 1] + 1];
	  primitives[index].n1.z = nbo[3 * ibo[3 * index + 1] + 2];
	  primitives[index].n2.x = nbo[3 * ibo[3 * index + 2] + 0];
	  primitives[index].n2.y = nbo[3 * ibo[3 * index + 2] + 1];
	  primitives[index].n2.z = nbo[3 * ibo[3 * index + 2] + 2];

  }
}

//__host__ __device__ float atomic(glm::vec3 barycentricCoord, float newDepth){
//  return -(barycentricCoord.x*tri.p0.z + barycentricCoord.y*tri.p1.z + barycentricCoord.z*tri.p2.z);
//}
__global__ void computeVertexNormal(triangle* primitives, int primitivesCount, fragment* depthbuffer, glm::vec2 resolution){

}

//TODO: Implement a rasterization method, such as scanline.
__global__ void rasterizationKernel(triangle* primitives, int primitivesCount, fragment* depthbuffer, glm::vec2 resolution, bool* edgeFlag, glm::vec3 eye, bool backCulling){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index < primitivesCount){

	  glm::vec3 normal =  primitives[index].n0;// glm::normalize(glm::cross(primitives[index].p1 - primitives[index].p0, primitives[index].p2 - primitives[index].p0));

	  //Back-face culling
	  if(backCulling == true){
		  if(glm::dot(eye, normal) > 0)
			  return;
	  }
	  glm::vec3 minPoint;
	  glm::vec3 maxPoint;


	  getAABBForTriangle(primitives[index], minPoint, maxPoint);

	  for(int j = max( (int)floor(minPoint.y)-1, 0); j < min( (int)ceil(maxPoint.y)+1, (int)resolution.y ); ++j){
		  for(int i = max( (int)floor(minPoint.x)-1, 0); i < min( (int)ceil(maxPoint.x)+1, (int)resolution.x); ++i){



			 glm::vec3 barycentricCoordSub;
			 for(int x = -1; x < 2; x++){
				for(int y = -1; y < 2; y++){ 
					barycentricCoordSub += calculateBarycentricCoordinate(primitives[index], glm::vec2(i + x, j + y));
				}
			 }

			glm::vec3 barycentricCoord = barycentricCoordSub / (float)9.0;


			
			 if(barycentricCoord.x < 0 || barycentricCoord.y < 0 || barycentricCoord.z < 0)
				 continue;

			//glm::vec3 barycentricCoord = calculateBarycentricCoordinate(primitives[index], glm::vec2(i, j));
			//glm::vec3 barycentricCoord1 = calculateBarycentricCoordinate(primitives[index], glm::vec2(i, j));
			//glm::vec3 barycentricCoord2 = calculateBarycentricCoordinate(primitives[index], glm::vec2(i, j + 1));
			//glm::vec3 barycentricCoord3 = calculateBarycentricCoordinate(primitives[index], glm::vec2(i + 1, j));
			//glm::vec3 barycentricCoord4 = calculateBarycentricCoordinate(primitives[index], glm::vec2(i + 1, j + 1));
			// if((barycentricCoord1.x < 0 || barycentricCoord1.y < 0 || barycentricCoord1.z < 0) &&
			//	(barycentricCoord2.x < 0 || barycentricCoord2.y < 0 || barycentricCoord2.z < 0) &&
			//	(barycentricCoord3.x < 0 || barycentricCoord3.y < 0 || barycentricCoord3.z < 0) &&
			//	(barycentricCoord4.x < 0 || barycentricCoord4.y < 0 || barycentricCoord4.z < 0))
			//	continue;

			 //if(edgeFlag[i + j * (int)resolution.x] == true)
				// continue;

			 float newDepth = -getZAtCoordinate(barycentricCoord, primitives[index]);
			 glm::vec3 newNormal;
			 newNormal.x = barycentricCoord.x * primitives[index].n0.x + barycentricCoord.y * primitives[index].n1.x + barycentricCoord.z * primitives[index].n2.x;
			 newNormal.y = barycentricCoord.x * primitives[index].n0.y + barycentricCoord.y * primitives[index].n1.y + barycentricCoord.z * primitives[index].n2.y;
			 newNormal.z = barycentricCoord.x * primitives[index].n0.z + barycentricCoord.y * primitives[index].n1.z + barycentricCoord.z * primitives[index].n2.z;

			 //newNormal.x = (barycentricCoord1.x * primitives[index].n0.x + barycentricCoord1.y * primitives[index].n1.x + barycentricCoord1.z * primitives[index].n2.x +
				//            barycentricCoord2.x * primitives[index].n0.x + barycentricCoord2.y * primitives[index].n1.x + barycentricCoord2.z * primitives[index].n2.x + 
				//		    barycentricCoord3.x * primitives[index].n0.x + barycentricCoord3.y * primitives[index].n1.x + barycentricCoord3.z * primitives[index].n2.x +
				//		    barycentricCoord4.x * primitives[index].n0.x + barycentricCoord4.y * primitives[index].n1.x + barycentricCoord4.z * primitives[index].n2.x) / 4;
			 //
			 //newNormal.y = (barycentricCoord1.x * primitives[index].n0.y + barycentricCoord1.y * primitives[index].n1.y + barycentricCoord1.z * primitives[index].n2.y +
				//            barycentricCoord2.x * primitives[index].n0.y + barycentricCoord2.y * primitives[index].n1.y + barycentricCoord2.z * primitives[index].n2.y + 
				//		    barycentricCoord3.x * primitives[index].n0.y + barycentricCoord3.y * primitives[index].n1.y + barycentricCoord3.z * primitives[index].n2.y +
				//		    barycentricCoord4.x * primitives[index].n0.y + barycentricCoord4.y * primitives[index].n1.y + barycentricCoord4.z * primitives[index].n2.y) / 4;
			
			 //newNormal.z = (barycentricCoord1.x * primitives[index].n0.z + barycentricCoord1.y * primitives[index].n1.z + barycentricCoord1.z * primitives[index].n2.z +
				//            barycentricCoord2.x * primitives[index].n0.z + barycentricCoord2.y * primitives[index].n1.z + barycentricCoord2.z * primitives[index].n2.z + 
				//		    barycentricCoord3.x * primitives[index].n0.z + barycentricCoord3.y * primitives[index].n1.z + barycentricCoord3.z * primitives[index].n2.z +
				//		    barycentricCoord4.x * primitives[index].n0.z + barycentricCoord4.y * primitives[index].n1.z + barycentricCoord4.z * primitives[index].n2.z) / 4;



			 float old = depthbuffer[i + j * (int)resolution.x].position.z;
			 float assumed;
			 do{
				assumed = old;

				if(assumed == depthbuffer[i + j * (int)resolution.x].position.z){
					//old = depthbuffer[i + j * (int)resolution.x].position.z;
					if(newDepth > depthbuffer[i + j * (int)resolution.x].position.z){
						depthbuffer[i + j * (int)resolution.x].position.z = newDepth;
						depthbuffer[i + j * (int)resolution.x].normal = newNormal;
						depthbuffer[i + j * (int)resolution.x].color.x = abs(newNormal.x);
						depthbuffer[i + j * (int)resolution.x].color.y = abs(newNormal.y);
						depthbuffer[i + j * (int)resolution.x].color.z = abs(newNormal.z);
					}
				}
				else{
					old =depthbuffer[i + j * (int)resolution.x].position.z;
				}
			 }
			 while(assumed != old);

			//if(abs(barycentricCoord.x) < 0.00001 || abs(barycentricCoord.y) < 0.00001 || abs(barycentricCoord.z) < 0.00001)
			//	 edgeFlag[i + j * (int)resolution.x] = true;
			 


			 //if(newDepth > depthbuffer[i + j * (int)resolution.x].position.z 
				// ){
			 //depthbuffer[i + j * (int)resolution.x].position.z = newDepth;
				//		depthbuffer[i + j * (int)resolution.x].normal = normal;
				//	}

		  }
	  }
  }
}



__global__ void stencilTest(fragment* depthbuffer, int startX, int startY, int endX, int endY){

}

__global__ void scissorTest(fragment* depthbuffer, int startX, int startY, int endX, int endY){

}

//TODO: Implement a fragment shader
__global__ void fragmentShadeKernel(fragment* depthbuffer, glm::vec2 resolution, glm::vec3 lightDir, bool scissorTest, glm::vec3 eye){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  //int inverseIndex = (resolution.x - x) + ((resolution.y - y) * resolution.x);
  if(x<=resolution.x && y<=resolution.y){

	  if(scissorTest == true){
		  if(x >= 300 && x <= 500 && y >=300 && y <= 500)
			  return;
	  }

	  float diffuseTerm = max(glm::dot(depthbuffer[index].normal, lightDir), (float)0.0);

	  int specularExp = 10;
	  glm::vec3 eveVector = (float)-1*glm::normalize(eye - depthbuffer[index].position);
	  glm::vec3 H = (eveVector + lightDir) / (float)2;
	  float specularTerm = max(pow(glm::dot(depthbuffer[index].normal, H), specularExp), (float)0.0);

	  //float colorR = depthbuffer[index].color.x * (diffuseTerm + specularTerm);
	  //float colorG = depthbuffer[index].color.y * (diffuseTerm + specularTerm);
	  //float colorB = depthbuffer[index].color.z * (diffuseTerm + specularTerm);


	  //depthbuffer[index].color.x = depthbuffer[index].color.x;
	  //depthbuffer[index].color.y = depthbuffer[index].color.y;
	  //depthbuffer[index].color.z = depthbuffer[index].color.z;

	  //depthbuffer[index].color.x = depthbuffer[index].position.z;
	  //depthbuffer[index].color.y = depthbuffer[index].position.z;
	  //depthbuffer[index].color.z = depthbuffer[index].position.z;

	  //depthbuffer[index].color.x = abs(depthbuffer[index].normal.x);
	  //depthbuffer[index].color.y = abs(depthbuffer[index].normal.y);
	  //depthbuffer[index].color.z = abs(depthbuffer[index].normal.z);

	  //depthbuffer[index].color.x = diffuseTerm * depthbuffer[index].color.x;
	  //depthbuffer[index].color.y = diffuseTerm * depthbuffer[index].color.y;
	  //depthbuffer[index].color.z = diffuseTerm * depthbuffer[index].color.z;

	  depthbuffer[index].color.x = specularTerm;
	  depthbuffer[index].color.y = specularTerm;
	  depthbuffer[index].color.z = specularTerm;
  }
}



//TODO:
__global__ void colorAlphaBlendingKernel(fragment* depthbuffer, glm::vec2 resolution, float alphaValue){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	if(x<=resolution.x && y<=resolution.y){

		if(depthbuffer[index].color != glm::vec3(0,0,0)){

			glm::vec3 destinationColor;
			int tempX = x / 50;
			int tempY = y / 50;
			if((tempX + tempY) % 2 == 0)
				destinationColor = glm::vec3(0,0,0);
			else
				destinationColor = glm::vec3(1,1,1);

			depthbuffer[index].color = depthbuffer[index].color * alphaValue + destinationColor * (1 - alphaValue);
		}
	}
}

__global__ void colorAdditiveBlendingKernel(fragment* depthbuffer, glm::vec2 resolution){

}

//Writes fragment colors to the framebuffer
__global__ void render(glm::vec2 resolution, fragment* depthbuffer, glm::vec3* framebuffer){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  int inverseIndex = (resolution.x - x) + ((resolution.y - y) * resolution.x);
  if(x<=resolution.x && y<=resolution.y){
    framebuffer[index] = depthbuffer[inverseIndex].color;
  }
}

// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRasterizeCore(uchar4* PBOpos, glm::vec2 resolution, float frame, float* vbo, int vbosize, float* cbo, int cbosize, int* ibo, int ibosize, 
	float* nbo, int nbosize, cudaMat4 shaderMatrix, int translateX, int translateY, glm::vec3 eye, glm::vec3 light, bool alphaBlend, float alphaValue, 
	bool backCulling, bool scissorTest){

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(resolution.x)/float(tileSize)), (int)ceil(float(resolution.y)/float(tileSize)));

  //set up framebuffer
  framebuffer = NULL;
  hipMalloc((void**)&framebuffer, (int)resolution.x*(int)resolution.y*sizeof(glm::vec3));
  
  edgeFlag = NULL;
  hipMalloc((void**)&edgeFlag, (int)resolution.x*(int)resolution.y*sizeof(bool));

  //set up depthbuffer
  depthbuffer = NULL;
  hipMalloc((void**)&depthbuffer, (int)resolution.x*(int)resolution.y*sizeof(fragment));

  //kernel launches to black out accumulated/unaccumlated pixel buffers and clear our scattering states
  clearImage<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, framebuffer, glm::vec3(0,0,0));
  
  fragment frag;
  frag.color = glm::vec3(0,0,0);
  frag.normal = glm::vec3(0,0,0);
  frag.position = glm::vec3(0,0,-10000);
  clearDepthBuffer<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer,frag);

  //------------------------------
  //memory stuff
  //------------------------------
  primitives = NULL;
  hipMalloc((void**)&primitives, (ibosize/3)*sizeof(triangle));

  device_ibo = NULL;
  hipMalloc((void**)&device_ibo, ibosize*sizeof(int));
  hipMemcpy( device_ibo, ibo, ibosize*sizeof(int), hipMemcpyHostToDevice);

  device_nbo = NULL;
  hipMalloc((void**)&device_nbo, nbosize*sizeof(float));
  hipMemcpy( device_nbo, nbo, nbosize*sizeof(float), hipMemcpyHostToDevice);

  device_vbo = NULL;
  hipMalloc((void**)&device_vbo, vbosize*sizeof(float));
  hipMemcpy( device_vbo, vbo, vbosize*sizeof(float), hipMemcpyHostToDevice);

  device_cbo = NULL;
  hipMalloc((void**)&device_cbo, cbosize*sizeof(float));
  hipMemcpy( device_cbo, cbo, cbosize*sizeof(float), hipMemcpyHostToDevice);



  tileSize = 32;
  int primitiveBlocks = ceil(((float)vbosize/3)/((float)tileSize));

  //------------------------------
  //vertex shader
  //------------------------------
  vertexShadeKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize, shaderMatrix, translateX, translateY);

  hipDeviceSynchronize();
  //------------------------------
  //primitive assembly
  //------------------------------
  primitiveBlocks = ceil(((float)ibosize/3)/((float)tileSize));
  primitiveAssemblyKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize, device_cbo, cbosize, device_ibo, ibosize, device_nbo, nbosize, primitives);

  hipDeviceSynchronize();
  //------------------------------
  //rasterization
  //------------------------------
  rasterizationKernel<<<primitiveBlocks, tileSize>>>(primitives, ibosize/3, depthbuffer, resolution, edgeFlag, eye, backCulling);

  hipDeviceSynchronize();
  //------------------------------
  //fragment shader
  //------------------------------
  fragmentShadeKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(depthbuffer, resolution, light, scissorTest, eye);

  hipDeviceSynchronize();

  //------------------------------
  //color blending
  //------------------------------
  if(alphaBlend){
	  colorAlphaBlendingKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(depthbuffer, resolution, alphaValue);
	  hipDeviceSynchronize();
  }
  //------------------------------
  //write fragments to framebuffer
  //------------------------------
  render<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer, framebuffer);
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, resolution, framebuffer);

  hipDeviceSynchronize();

  kernelCleanup();

  checkCUDAError("Kernel failed!");
}

void kernelCleanup(){
  hipFree( primitives );
  hipFree( device_vbo );
  hipFree( device_cbo );
  hipFree( device_ibo );
  hipFree( device_nbo );
  hipFree( framebuffer );
  hipFree( depthbuffer );
  hipFree( edgeFlag );
}

