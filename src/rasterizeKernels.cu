#include "hip/hip_runtime.h"
// CIS565 CUDA Rasterizer: A simple rasterization pipeline for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/random.h>
#include "rasterizeKernels.h"
#include "rasterizeTools.h"

glm::vec3* framebuffer;
fragment* depthbuffer;
float* device_vbo;
float* device_cbo;
float* device_nbo;
int* device_ibo;
triangle* primitives;
//camera info
glm::vec3 up(0, 1, 0);
float fovy = 50;
float zNear = 0.01;
float zFar = 1000;
//light info
light* lights;
int lightsize = 4;
//switch
bool isAntiAlias = false;

using namespace std;

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//Handy dandy little hashing function that provides seeds for random number generation
__host__ __device__ unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Writes a given fragment to a fragment buffer at a given location
__host__ __device__ void writeToDepthbuffer(int x, int y, fragment frag, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    depthbuffer[index] = frag;
  }
}

//Reads a fragment from a given location in a fragment buffer
__host__ __device__ fragment getFromDepthbuffer(int x, int y, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return depthbuffer[index];
  }else{
    fragment f;
    return f;
  }
}

//Writes a given pixel to a pixel buffer at a given location
__host__ __device__ void writeToFramebuffer(int x, int y, glm::vec3 value, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    framebuffer[index] = value;
  }
}

//Reads a pixel from a pixel buffer at a given location
__host__ __device__ glm::vec3 getFromFramebuffer(int x, int y, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return framebuffer[index];
  }else{
    return glm::vec3(0,0,0);
  }
}

//Kernel that clears a given pixel buffer with a given color
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image, glm::vec3 color){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = color;
    }
}

//Kernel that clears a given fragment buffer with a given fragment
__global__ void clearDepthBuffer(glm::vec2 resolution, fragment* buffer, fragment frag){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      fragment f = frag;
      f.position.x = x;
      f.position.y = y;
      buffer[index] = f;
	  buffer[index].z = -FLT_MAX;
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;      
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

__host__ __device__ glm::vec3 transformPos(glm::vec3 v, glm::mat4 matrix, glm::vec2 resolution) {
	glm::vec4 v4(v, 1);
	v4 = matrix * v4;
	// perspective division
	v4.x = v4.x/v4.w;
	v4.y = v4.y/v4.w;
	v4.z = v4.z/v4.w;
	// viewport transform
	v4.x = resolution.x/2 * (v4.x+1);
	v4.y = resolution.y/2 * (v4.y+1);
	v4.z = -0.5 * v4.z + 0.5;

	return glm::vec3(v4);
}

__global__ void transformVertices(float* vbo, int vbosize, glm::mat4 modelMatrix) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(index<vbosize/3){
		glm::vec4 v(vbo[index*3], vbo[index*3+1], vbo[index*3+2], 1);
		v = modelMatrix * v;
		vbo[index*3] = v.x;
		vbo[index*3+1] = v.y;
		vbo[index*3+2] = v.z;
	}
}

__global__ void transformNormals(float* nbo, int nbosize, glm::mat4 modelMatrix) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(index<nbosize/3){
		glm::vec4 n(nbo[index*3], nbo[index*3+1], nbo[index*3+2], 0);
		n = modelMatrix * n;
		nbo[index*3] = n.x;
		nbo[index*3+1] = n.y;
		nbo[index*3+2] = n.z;
	}
}

//TODO: Implement a vertex shader
__global__ void vertexShadeKernel(float* vbo, int vbosize, glm::mat4 cameraMatrix, glm::vec2 resolution){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index<vbosize/3){
	   glm::vec3 v(vbo[index*3], vbo[index*3+1], vbo[index*3+2]);
	  v = transformPos(v, cameraMatrix, resolution);
	  vbo[index*3] = v.x;
	  vbo[index*3+1] = v.y;
	  vbo[index*3+2] = v.z;
  }
}
__global__ void updatePrimitiveKernel(float* vbo, int vbosize, int* ibo, int ibosize, triangle* primitives){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int primitivesCount = ibosize/3;
  if(index<primitivesCount){
	  int v0 = ibo[index*3];
	  int v1 = ibo[index*3+1];
	  int v2 = ibo[index*3+2];
	  primitives[index].pt0 = glm::vec3(vbo[v0*3], vbo[v0*3+1], vbo[v0*3+2]);
	  primitives[index].pt1 = glm::vec3(vbo[v1*3], vbo[v1*3+1], vbo[v1*3+2]);
	  primitives[index].pt2 = glm::vec3(vbo[v2*3], vbo[v2*3+1], vbo[v2*3+2]);
  }
}

//TODO: Implement primative assembly
__global__ void primitiveAssemblyKernel(float* vbo, int vbosize, float* cbo, int cbosize, int* ibo, int ibosize, float* nbo, int nbosize, triangle* primitives){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int primitivesCount = ibosize/3;
  if(index<primitivesCount){
	  int v0 = ibo[index*3];
	  int v1 = ibo[index*3+1];
	  int v2 = ibo[index*3+2];
	  glm::vec3 p0(vbo[v0*3], vbo[v0*3+1], vbo[v0*3+2]);
	  glm::vec3 p1(vbo[v1*3], vbo[v1*3+1], vbo[v1*3+2]);
	  glm::vec3 p2(vbo[v2*3], vbo[v2*3+1], vbo[v2*3+2]);
	  glm::vec3 c0(cbo[0], cbo[1], cbo[2]);
	  glm::vec3 c1(cbo[0], cbo[1], cbo[2]);
	  glm::vec3 c2(cbo[0], cbo[1], cbo[2]);
	 /* glm::vec3 c0(cbo[v0*3], cbo[v0*3+1], cbo[v0*3+2]);
	  glm::vec3 c1(cbo[v1*3], cbo[v1*3+1], cbo[v1*3+2]);
	  glm::vec3 c2(cbo[v2*3], cbo[v2*3+1], cbo[v2*3+2]);*/
	  glm::vec3 n0(nbo[v0*3], nbo[v0*3+1], nbo[v0*3+2]);
	  glm::vec3 n1(nbo[v1*3], nbo[v1*3+1], nbo[v1*3+2]);
	  glm::vec3 n2(nbo[v2*3], nbo[v2*3+1], nbo[v2*3+2]);
	  primitives[index] = triangle(p0, p1, p2, c0, c1, c2, n0, n1, n2);
  }
}
//Anti alias converge
__global__ void converge(glm::vec2 resolution, fragment* buffer, fragment* antialiasBuffer){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);
	if(x<=resolution.x && y<=resolution.y){
		glm::vec3 c(0,0,0);
		for(int i=x*2;i<x*2+2;i++)
		for(int j=y*2;j<y*2+2;j++)
			c+=antialiasBuffer[i+j*(int)resolution.x*2].color;
		buffer[index].color=c*0.25f;

		//buffer[index].color=antialiasBuffer[x+y*(int)resolution.x].color;
	}
}
//TODO: Implement a rasterization method, such as scanline.
__device__ glm::vec3 getScanlineIntersection(glm::vec3 v1, glm::vec3 v2, float y) {
	float t = (y-v1.y)/(v2.y-v1.y);
	return glm::vec3(t*v2.x + (1-t)*v1.x, y, t*v2.z + (1-t)*v1.z);
}

__global__ void rasterizationKernel(triangle* primitives, int primitivesCount, fragment* depthbuffer, glm::vec2 resolution){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < primitivesCount) {
		triangle prim = primitives[index];
		/*float topy = min(min(prim.pt0.y, prim.pt1.y), prim.pt2.y);
		float boty = max(max(prim.pt0.y, prim.pt1.y), prim.pt2.y);
		int top = max((int)floor(topy), 0);
		int bot = min((int)ceil(boty), (int)resolution.y);*/
		glm::vec3 minPoint, maxPoint;
		getAABBForTriangle(prim, minPoint, maxPoint);

		for (int y = minPoint.y; y < maxPoint.y; ++y) {
			float dy0 = prim.pt0.y - y;
			float dy1 = prim.pt1.y - y;
			float dy2 = prim.pt2.y - y;
			int onPositiveSide = (int)(dy0>=0) + (int)(dy1>=0) + (int)(dy2>=0);
			int onNegativeSide = (int)(dy0<=0) + (int)(dy1<=0) + (int)(dy2<=0);

			glm::vec3 intersection1, intersection2;
			if (onPositiveSide == 3 || onNegativeSide == 3) {
				if (dy0 == 0) {
					intersection1 = prim.pt0;
					intersection2 = prim.pt0;
				}
				else if (dy1 == 0) {
					intersection1 = prim.pt1;
					intersection2 = prim.pt1;
				}
				else if (dy2 == 0) {
					intersection1 = prim.pt2;
					intersection2 = prim.pt2;
				}
			}
			else if (onPositiveSide == 2 && onNegativeSide == 2) { // one vertex is on the scanline
															// doesn't really happen due to the floating point error
				if (dy0 == 0) {
					intersection1 = prim.pt0;
					intersection2 = getScanlineIntersection(prim.pt1, prim.pt2, y);
				}
				else if (dy1 == 0) {
					intersection1 = prim.pt1;
					intersection2 = getScanlineIntersection(prim.pt0, prim.pt2, y);
				}
				else { // dy2 == 0
					intersection1 = prim.pt2;
					intersection2 = getScanlineIntersection(prim.pt1, prim.pt0, y);
				}
			}
			else if (onPositiveSide == 2) {
				if (dy0 < 0) {
					intersection1 = getScanlineIntersection(prim.pt0, prim.pt1, y);
					intersection2 = getScanlineIntersection(prim.pt0, prim.pt2, y);
				}
				else if (dy1 < 0) {
					intersection1 = getScanlineIntersection(prim.pt1, prim.pt0, y);
					intersection2 = getScanlineIntersection(prim.pt1, prim.pt2, y);
				}
				else { // dy2 < 0
					intersection1 = getScanlineIntersection(prim.pt2, prim.pt0, y);
					intersection2 = getScanlineIntersection(prim.pt2, prim.pt1, y);
				}
			}
			else { // onNegativeSide == 2
				if (dy0 > 0) {
					intersection1 = getScanlineIntersection(prim.pt0, prim.pt1, y);
					intersection2 = getScanlineIntersection(prim.pt0, prim.pt2, y);
				}
				else if (dy1 > 0) {
					intersection1 = getScanlineIntersection(prim.pt1, prim.pt0, y);
					intersection2 = getScanlineIntersection(prim.pt1, prim.pt2, y);
				}
				else { // dy2 > 0
					intersection1 = getScanlineIntersection(prim.pt2, prim.pt0, y);
					intersection2 = getScanlineIntersection(prim.pt2, prim.pt1, y);
				}
			}

			// make sure intersection1's x value is less than intersection2's
			if (intersection2.x < intersection1.x) {
				glm::vec3 temp = intersection1;
				intersection1 = intersection2;
				intersection2 = temp;
			}

			int left = min((int)(resolution.x)-1,max(0, (int)floor(intersection1.x)));
			int right = min((int)(resolution.x-1),max(0, (int)floor(intersection2.x)));
			for (int x = left; x <= right; ++x) {//fill in
				int pixelIndex = (resolution.x-1-x) + (resolution.y-1-y) * resolution.x;
				float t = (x-intersection1.x)/(intersection2.x-intersection1.x);
				glm::vec3 point = t*intersection2 + (1-t)*intersection1;
						
				if (point.z > depthbuffer[pixelIndex].z) {
					glm::vec3 bc = calculateBarycentricCoordinate(prim, glm::vec2(point.x, point.y));
					depthbuffer[pixelIndex].color = prim.c0 * bc.x + prim.c1 * bc.y + prim.c2 * bc.z;//glm::vec3(1,0,0) * bc.x + glm::vec3(0,1,0) * bc.y + glm::vec3(0,0,1) * bc.z;//
					depthbuffer[pixelIndex].normal = glm::normalize(prim.n0 * bc.x + prim.n1 * bc.y + prim.n2 * bc.z);
					depthbuffer[pixelIndex].position = prim.p0 * bc.x + prim.p1 * bc.y + prim.p2 * bc.z;
					depthbuffer[pixelIndex].z = point.z;
				}
						
			}
		}
			
	}
}


//TODO: Implement a fragment shader
__global__ void fragmentShadeKernel(fragment* depthbuffer, glm::vec2 resolution, glm::vec3 eye, light* lights, int lightsize){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  if(x<=resolution.x && y<=resolution.y){
	  glm::vec3 diffuseColor(0);
		glm::vec3 specularColor(0);
		float ks = 0;
		if (glm::distance(depthbuffer[index].color, glm::vec3(245.0/255.0, 222.0/255.0, 179.0/255.0)) > 0.1) {
			ks = 0.3;
		}
		glm::vec3 norm =  depthbuffer[index].normal;
		glm::vec3 pos = depthbuffer[index].position;
		for (int i=0; i<lightsize; ++i) {
			//diffuse component
			glm::vec3 lightDir = glm::normalize(glm::vec3(lights[i].pos - pos));
			float diffuseTerm = glm::clamp(glm::dot(lightDir, norm), 0.0f, 1.0f);
			diffuseColor += diffuseTerm * lights[i].color;

			//specular component
			if (ks > 0.0001) {
				glm::vec3 LR; // reflected light direction
				if (glm::length(lightDir - norm) < 0.0001) {
					LR = norm;
				}
				else if (abs(glm::dot(lightDir, norm)) < 0.0001) {
					LR = -lightDir;
				}
				else {
					LR = glm::normalize(-lightDir - 2.0f * glm::dot(-lightDir, norm) * norm);
				}
				float specularTerm = min(1.0f, pow(max(0.0f, glm::dot(LR, glm::normalize(eye - pos))), 20.0f));
				specularColor += specularTerm * glm::vec3(1.0f);
			}
		}
		depthbuffer[index].color = diffuseColor * depthbuffer[index].color + ks * specularColor;

		//set background color
		if (depthbuffer[index].z == -FLT_MAX) {
			depthbuffer[index].color = glm::vec3(0.5, 0.5, 0.5);
		}
  }
}

//Writes fragment colors to the framebuffer
__global__ void render(glm::vec2 resolution, fragment* depthbuffer, glm::vec3* framebuffer){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if(x<=resolution.x && y<=resolution.y){
    framebuffer[index] = depthbuffer[index].color;
  }
}
void initLights() {
	light l1(glm::vec3(1.0, 1.0, 1.0), glm::vec3(4, -4, 4));
	light l2(glm::vec3(0.0, 0.0, 0.0), glm::vec3(4, -9, 4));
	light l3(glm::vec3(0.0, 0.0, 0.0), glm::vec3(0, -10, -5));
	light l4(glm::vec3(0.0, 0.0, 0.0), glm::vec3(0, -9, 0));
	light* cpulights = new light[lightsize];
	cpulights[0] = l1;
	cpulights[1] = l2;
	cpulights[2] = l3;
	cpulights[3] = l4;
	
	checkCUDAError("Kernel failed!");
	hipMalloc((void**)&lights, lightsize*sizeof(light));
	checkCUDAError("Kernel failed!");
	hipMemcpy(lights, cpulights, lightsize*sizeof(light), hipMemcpyHostToDevice);
	checkCUDAError("Kernel failed!");

	delete [] cpulights;
}
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRasterizeCore(uchar4* PBOpos, glm::vec2 resolution, glm::mat4 rotation,float frame, float* vbo, int vbosize, float* cbo, int cbosize, int* ibo, int ibosize, float* nbo, int nbosize, glm::vec3 eye, glm::vec3 center){

  // set up crucial magic
  int tileSize = 32;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(resolution.x)/float(tileSize)), (int)ceil(float(resolution.y)/float(tileSize)));

  //set up framebuffer
  framebuffer = NULL;
  hipMalloc((void**)&framebuffer, (int)resolution.x*(int)resolution.y*sizeof(glm::vec3));
  
  //set up depthbuffer
  depthbuffer = NULL;
  hipMalloc((void**)&depthbuffer, (int)resolution.x*(int)resolution.y*sizeof(fragment));

  //kernel launches to black out accumulated/unaccumlated pixel buffers and clear our scattering states
  clearImage<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, framebuffer, glm::vec3(0,0,0));
  
  fragment frag;
  frag.color = glm::vec3(0,0,0);
  frag.normal = glm::vec3(0,0,0);
  frag.position = glm::vec3(0,0,-10000);
  clearDepthBuffer<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer,frag);

  //------------------------------
  //memory stuff
  //------------------------------
  primitives = NULL;
  hipMalloc((void**)&primitives, (ibosize/3)*sizeof(triangle));

  device_ibo = NULL;
  hipMalloc((void**)&device_ibo, ibosize*sizeof(int));
  hipMemcpy( device_ibo, ibo, ibosize*sizeof(int), hipMemcpyHostToDevice);

  device_vbo = NULL;
  hipMalloc((void**)&device_vbo, vbosize*sizeof(float));
  hipMemcpy( device_vbo, vbo, vbosize*sizeof(float), hipMemcpyHostToDevice);

  device_cbo = NULL;
  hipMalloc((void**)&device_cbo, cbosize*sizeof(float));
  hipMemcpy( device_cbo, cbo, cbosize*sizeof(float), hipMemcpyHostToDevice);

  device_nbo = NULL;
  hipMalloc((void**)&device_nbo, nbosize*sizeof(float));
  hipMemcpy( device_nbo, nbo, nbosize*sizeof(float), hipMemcpyHostToDevice);
  tileSize = 32;
  //------------------------------
  //camera set up
  //------------------------------
  glm::vec2 resolution2(resolution.x*2,resolution.y*2);
  glm::vec2 reso = isAntiAlias?resolution2: resolution;
  glm::mat4 perspMatrix = glm::perspective(fovy, reso.x/reso.y, zNear, zFar);
  glm::mat4 lookatMatrix = glm::lookAt(eye, center, up);
  glm::mat4 cameraMatrix = perspMatrix * lookatMatrix;

  initLights();
  //------------------------------
  //vertex shader
  //------------------------------
	int primitiveBlocks = ceil(((float)vbosize/3)/((float)tileSize));
	transformVertices<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize, rotation);
	transformNormals<<<primitiveBlocks, tileSize>>>(device_nbo, nbosize, rotation);
	primitiveBlocks = ceil(((float)vbosize/3)/((float)tileSize));
	vertexShadeKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize, cameraMatrix, reso);

  hipDeviceSynchronize();
  //------------------------------
  //primitive assembly
  //------------------------------
  primitiveBlocks = ceil(((float)ibosize/3)/((float)tileSize));
  primitiveAssemblyKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize, device_cbo, cbosize, device_ibo, ibosize, device_nbo, nbosize, primitives);

  hipDeviceSynchronize();
  //----------------------------
  primitiveBlocks = ceil(((float)ibosize/3)/((float)tileSize));
  updatePrimitiveKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize, device_ibo, ibosize, primitives);

  hipDeviceSynchronize();
  //------------------------------
  //rasterization
  //------------------------------
  if(isAntiAlias){
	fragment* anti_alias_depthbuffer = NULL;
	
	dim3 fullBlocksPerGrid2((int)ceil(float(resolution2.x)/float(tileSize)), (int)ceil(float(resolution2.y)/float(tileSize)));
	dim3 threadsPerBlock2(tileSize,tileSize);
	hipMalloc((void**)&anti_alias_depthbuffer, 4*(int)resolution.x*(int)resolution.y*sizeof(fragment));
	clearDepthBuffer<<<fullBlocksPerGrid2, threadsPerBlock2>>>(resolution2, anti_alias_depthbuffer,frag);
	hipDeviceSynchronize();
	rasterizationKernel<<<primitiveBlocks, tileSize>>>(primitives, ibosize/3, anti_alias_depthbuffer, resolution2);

	fragmentShadeKernel<<<fullBlocksPerGrid2, threadsPerBlock2>>>(anti_alias_depthbuffer, resolution2, eye, lights, lightsize);
	converge<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer, anti_alias_depthbuffer);
	hipDeviceSynchronize();
	hipFree(anti_alias_depthbuffer);
	  
  }
  else{
	rasterizationKernel<<<primitiveBlocks, tileSize>>>(primitives, ibosize/3, depthbuffer, resolution);
	hipDeviceSynchronize();
	
	fragmentShadeKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(depthbuffer, resolution, eye, lights, lightsize);
	hipDeviceSynchronize();
  }
  //------------------------------
  //fragment shader
  //------------------------------
  
  //------------------------------
  //write fragments to framebuffer
  //------------------------------
  render<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer, framebuffer);
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, resolution, framebuffer);

  hipDeviceSynchronize();

  kernelCleanup();

  checkCUDAError("Kernel failed!");
}

void kernelCleanup(){
  hipFree( primitives );
  hipFree( device_vbo );
  hipFree( device_cbo );
  hipFree( device_ibo );
  hipFree( device_nbo );
  hipFree( framebuffer );
  hipFree( depthbuffer );
}

