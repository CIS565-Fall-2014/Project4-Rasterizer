#include "hip/hip_runtime.h"
// CIS565 CUDA Rasterizer: A simple rasterization pipeline for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/random.h>
#include "rasterizeKernels.h"
#include "rasterizeTools.h"

glm::vec3 *framebuffer;
fragment *depthbuffer;
float *device_vbo;
float *device_cbo;
int *device_ibo;
float *device_nbo;
triangle* primitives;
float *device_vbo_window_coords;

const float EMPTY_BUFFER_DEPTH = -10000.0f;

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//Handy dandy little hashing function that provides seeds for random number generation
__host__ __device__ unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Writes a given fragment to a fragment buffer at a given location
__host__ __device__ void writeToDepthbuffer(int x, int y, fragment frag, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    depthbuffer[index] = frag;
  }
}

//Reads a fragment from a given location in a fragment buffer
__host__ __device__ fragment getFromDepthbuffer(int x, int y, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return depthbuffer[index];
  }else{
    fragment f;
    return f;
  }
}

//Writes a given pixel to a pixel buffer at a given location
__host__ __device__ void writeToFramebuffer(int x, int y, glm::vec3 value, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    framebuffer[index] = value;
  }
}

//Reads a pixel from a pixel buffer at a given location
__host__ __device__ glm::vec3 getFromFramebuffer(int x, int y, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return framebuffer[index];
  }else{
    return glm::vec3(0,0,0);
  }
}

//Kernel that clears a given pixel buffer with a given color
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image, glm::vec3 color){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = color;
    }
}

//Kernel that clears a given fragment buffer with a given fragment
__global__ void clearDepthBuffer(glm::vec2 resolution, fragment* buffer, fragment frag){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      fragment f = frag;
      f.position.x = x;
      f.position.y = y;
      buffer[index] = f;
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;      
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}


/*********** DANNY'S PRIMARY CONTRIBUTION - START ***********/

// Convert vertices from object-space coordinates to window coordinates.
__global__
void vertexShadeKernel( float *vbo,
						int vbosize,
						glm::mat4 mvp_matrix,
						glm::vec2 resolution,
						float *vbo_window_coords )
{
	int index = ( blockIdx.x * blockDim.x ) + threadIdx.x;

	// Divide by 3 because each vertex has 3 components (x, y, and z).
	if ( index < vbosize / 3 ) {
		int vbo_index = index * 3;

		// Create point to transform.
		glm::vec4 v( vbo[vbo_index + 0], vbo[vbo_index + 1], vbo[vbo_index + 2], 1.0f );

		// Transform point from object-space to clip-space by multiplying by the composite model, view, projection matrices.
		glm::vec4 vt = mvp_matrix * v;

		// Transform point to NDC-space by dividing x-, y-, and z-components by w-component (perspective division).
		// [-1, 1].
		glm::vec3 v_ndc( vt.x / vt.w, vt.y / vt.w, vt.z / vt.w );

		// Transform x and y range from [-1, 1] to [0, 1].
		glm::vec2 v_remapped( ( v_ndc.x + 1.0f ) / 2.0f, ( v_ndc.y + 1.0f ) / 2.0f );

		// Transform x- and y-coordinates to window-space.
		glm::vec2 v_window( v_remapped.x * resolution.x, v_remapped.y * resolution.y );

		// Save transformed vertices.
		vbo_window_coords[vbo_index + 0] = v_window.x;
		vbo_window_coords[vbo_index + 1] = v_window.y;
		vbo_window_coords[vbo_index + 2] = v_ndc.z;
	}
}


template<typename T>
__host__
__device__
void simpleSwap( T &f1, T &f2 )
{
	T tmp = f1;
	f1 = f2;
	f2 = tmp;
}


// Construct primitives from vertices.
__global__
void primitiveAssemblyKernel( float *vbo, int vbosize,
							  float *cbo, int cbosize,
							  int *ibo, int ibosize,
							  float *nbo, int nbosize,
							  float *vbo_window_coords,
							  triangle *primitives )
{
	int index = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	int primitivesCount = ibosize / 3;
	if ( index < primitivesCount ) {
		// Get indices of triangle vertices.
		int ibo_index = index * 3;
		int i0 = ibo[ibo_index + 0];
		int i1 = ibo[ibo_index + 1];
		int i2 = ibo[ibo_index + 2];

		// Get positions of triangle vertices.
		int v0_index = i0 * 3;
		int v1_index = i1 * 3;
		int v2_index = i2 * 3;

		// Get screen-space positions of triangle vertices.
		glm::vec3 ssp0( vbo_window_coords[v0_index + 0], vbo_window_coords[v0_index + 1], vbo_window_coords[v0_index + 2] );
		glm::vec3 ssp1( vbo_window_coords[v1_index + 0], vbo_window_coords[v1_index + 1], vbo_window_coords[v1_index + 2] );
		glm::vec3 ssp2( vbo_window_coords[v2_index + 0], vbo_window_coords[v2_index + 1], vbo_window_coords[v2_index + 2] );

		// Check if triangle is visible.
		glm::vec3 backface_check = glm::cross( ssp1 - ssp0, ssp2 - ssp0 );
		if ( backface_check.z < 0.0f ) {
			triangle tri;
			tri.is_visible = false;
			primitives[index] = tri;
			return;
		}

		// Get positions of triangle vertices.
		glm::vec3 p0( vbo[v0_index + 0], vbo[v0_index + 1], vbo[v0_index + 2] );
		glm::vec3 p1( vbo[v1_index + 0], vbo[v1_index + 1], vbo[v1_index + 2] );
		glm::vec3 p2( vbo[v2_index + 0], vbo[v2_index + 1], vbo[v2_index + 2] );

		// Get colors of triangle vertices.
		int c0_index = ( i0 % 3 ) * 3;
		int c1_index = ( i1 % 3 ) * 3;
		int c2_index = ( i2 % 3 ) * 3;
		glm::vec3 c0( cbo[c0_index + 0], cbo[c0_index + 1], cbo[c0_index + 2] );
		glm::vec3 c1( cbo[c1_index + 0], cbo[c1_index + 1], cbo[c1_index + 2] );
		glm::vec3 c2( cbo[c2_index + 0], cbo[c2_index + 1], cbo[c2_index + 2] );

		// Get normals of triangle vertices.
		glm::vec3 n0( nbo[v0_index + 0], nbo[v0_index + 1], nbo[v0_index + 2] );
		glm::vec3 n1( nbo[v1_index + 0], nbo[v1_index + 1], nbo[v1_index + 2] );
		glm::vec3 n2( nbo[v2_index + 0], nbo[v2_index + 1], nbo[v2_index + 2] );

		// Set triangle.
		primitives[index] = triangle( p0, p1, p2,
									  ssp0, ssp1, ssp2,
									  c0, c1, c2,
									  n0, n1, n2 );
	}
}


//__device__ glm::vec3 getScanlineIntersection(glm::vec3 v1, glm::vec3 v2, float y) {
//	float t = (y-v1.y)/(v2.y-v1.y);
//	return glm::vec3(t*v2.x + (1-t)*v1.x, y, t*v2.z + (1-t)*v1.z);
//}
//
//__host__
//__device__
//glm::vec3 computePoint


// Scanline rasterization per triangle.
// Thanks: http://graphics.stanford.edu/courses/cs248-08/scan/scan1.html
__global__
void rasterizationKernel( triangle *primitives,
						  int primitivesCount,
						  fragment *depthbuffer,
						  glm::vec2 resolution )
{
	int index = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	if ( index < primitivesCount ) {
		triangle tri = primitives[index];

		// Only rasterize current triangle if triangle is visible (determined in primitive assembly stage).
		if ( !tri.is_visible ) {
			return;
		}

		// Get screen-space vertices for current triangle.
		glm::vec3 v1 = tri.ssp0;
		glm::vec3 v2 = tri.ssp1;
		glm::vec3 v3 = tri.ssp2;

		// Sort triangle vertices in ascending order by screen-space y-coordinate.
		if ( v1.y > v2.y ) {
			simpleSwap( v1, v2 );
		}
		if ( v1.y > v3.y ) {
			simpleSwap( v1, v3 );
		}
		if ( v2.y > v3.y ) {
			simpleSwap( v2, v3 );
		}

		// If triangle vertices have same y-coordinate, then sort in ascending order by screen-space x-coordinate.
		if ( v1.y == v2.y && v1.x > v2.x ) {
			simpleSwap( v1, v2 );
		}
		if ( v2.y == v3.y && v2.x > v3.x  ) {
			simpleSwap( v2, v3 );
		}

		int y_bot = ceil( v1.y );
		int y_mid = ceil( v2.y );
		int y_top = ceil( v3.y );

		edge e1, e2, e3, l, r;
		e1.setEdge( v1, v3, y_bot );
		e2.setEdge( v1, v2, y_bot );
		e3.setEdge( v2, v3, y_mid );

		// Set left and right edges based on x-values.
		if ( v1.x < v2.x ) {
			l = e1;
			r = e2;
		}
		else {
			l = e2;
			r = e1;
		}

		// Loop through scanlines covered by the triangle.
		for ( int y = y_bot; y < y_top - 1; ++y ) {
			// Update edge if scanline has reached the mid-y triangle point.
			if ( y >= y_mid ) {				
				if ( v1.x < v2.x ) {
					r = e3;
				}
				else {
					l = e3;
				}
			}

			int lx = ceil( l.x );
			int rx = ceil( r.x );

			for ( int x = lx; x < rx - 1; ++x ) {
				if ( x > 0 && x < resolution.x && y > 0 && y < resolution.y ) {

					// TODO: current_z is computed WRT triangle in object-space. I think it should be computed WRT triangle in camera-space.

					// Compute Barycentric coordinates of current fragment in screen-space triangle.
					glm::vec3 barycentric_coordinates = calculateBarycentricCoordinate( tri.ssp0, tri.ssp1, tri.ssp2, glm::vec2( x, y ) );
					float current_z = getZAtCoordinate( barycentric_coordinates, tri.p0, tri.p1, tri.p2 );

					fragment buffer_fragment = getFromDepthbuffer( x, y, depthbuffer, resolution );
					float buffer_z = buffer_fragment.position.z;

					// Update depth buffer.
					if ( current_z > buffer_z ) {
						fragment f;
						f.color = ( tri.c0 * barycentric_coordinates.x ) + ( tri.c1 * barycentric_coordinates.y ) + ( tri.c2 * barycentric_coordinates.z );
						f.normal = glm::normalize( ( tri.n0 * barycentric_coordinates.x ) + ( tri.n1 * barycentric_coordinates.y ) + ( tri.n2 * barycentric_coordinates.z ) );
						f.position = ( tri.p0 * barycentric_coordinates.x ) + ( tri.p1 * barycentric_coordinates.y ) + ( tri.p2 * barycentric_coordinates.z );
						writeToDepthbuffer( x, y, f, depthbuffer, resolution );
					}
				}
			}

			l.x += l.dxdy;
			r.x += r.dxdy;
		}
	}
}


// Compute light interaction with fragments.
// Write fragment colors to frame buffer.
__global__
void fragmentShadeKernel( fragment *depthbuffer,
						  glm::vec2 resolution )
{
	int x = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	int y = ( blockIdx.y * blockDim.y ) + threadIdx.y;
	int index = x + ( y * resolution.x );
	if ( x <= resolution.x && y <= resolution.y ) {

		// TODO.

	}
}

/*********** DANNY'S PRIMARY CONTRIBUTION - END ***********/


//Writes fragment colors to the framebuffer
__global__ void render(glm::vec2 resolution, fragment* depthbuffer, glm::vec3* framebuffer){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if(x<=resolution.x && y<=resolution.y){
    framebuffer[index] = depthbuffer[index].color;
  }
}

// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRasterizeCore( uchar4 *PBOpos,
						float frame,
						float *vbo, int vbosize,
						float *cbo, int cbosize,
						int *ibo, int ibosize,
						float *nbo, int nbosize,
						simpleCamera camera )
{
	// set up crucial magic
	int tileSize = 8;
	dim3 threadsPerBlock( tileSize,
						  tileSize );
	dim3 fullBlocksPerGrid( ( int )ceil( ( float )camera.resolution.x / ( float )tileSize ),
							( int )ceil( ( float )camera.resolution.y / ( float )tileSize ) );

	// set up framebuffer
	framebuffer = NULL;
	hipMalloc( ( void** )&framebuffer,
				( int )camera.resolution.x * ( int )camera.resolution.y * sizeof( glm::vec3 ) );
  
	// set up depthbuffer
	depthbuffer = NULL;
	hipMalloc( ( void** )&depthbuffer,
				( int )camera.resolution.x * ( int )camera.resolution.y * sizeof( fragment ) );

	// kernel launches to black out accumulated/unaccumlated pixel buffers and clear our scattering states
	clearImage<<< fullBlocksPerGrid, threadsPerBlock >>>( camera.resolution,
														  framebuffer,
														  glm::vec3( 0.0f, 0.0f, 0.0f ) );
  
	fragment frag;
	frag.color = glm::vec3( 0.0f, 0.0f, 0.0f );
	frag.normal = glm::vec3( 0.0f, 0.0f, 0.0f );
	frag.position = glm::vec3( 0.0f, 0.0f, EMPTY_BUFFER_DEPTH );
	clearDepthBuffer<<< fullBlocksPerGrid, threadsPerBlock >>>( camera.resolution,
																depthbuffer,
																frag );

	//------------------------------
	// memory stuff
	//------------------------------
	primitives = NULL;
	hipMalloc( ( void** )&primitives,
				( ibosize / 3 ) * sizeof( triangle ) );

	device_ibo = NULL;
	hipMalloc( ( void** )&device_ibo,
				ibosize * sizeof( int ) );
	hipMemcpy( device_ibo,
				ibo,
				ibosize * sizeof( int ),
				hipMemcpyHostToDevice );

	device_vbo = NULL;
	hipMalloc( ( void** )&device_vbo,
				vbosize * sizeof( float ) );
	hipMemcpy( device_vbo,
				vbo,
				vbosize * sizeof( float ),
				hipMemcpyHostToDevice );

	device_vbo_window_coords = NULL;
	hipMalloc( ( void** )&device_vbo_window_coords,
				vbosize * sizeof( float ) );

	device_cbo = NULL;
	hipMalloc( ( void** )&device_cbo,
				cbosize * sizeof( float ) );
	hipMemcpy( device_cbo,
				cbo,
				cbosize * sizeof( float ),
				hipMemcpyHostToDevice );

	device_nbo = NULL;
	hipMalloc( ( void** )&device_nbo,
				nbosize * sizeof( float ) );
	hipMemcpy( device_nbo,
				nbo,
				nbosize * sizeof( float ),
				hipMemcpyHostToDevice );

	tileSize = 32;
	int primitiveBlocks = ceil( ( ( float )vbosize / 3 ) / ( ( float )tileSize ) );

	//------------------------------
	// vertex shader
	//------------------------------

	// Define model matrix.
	// Transforms from object-space to world-space.
	glm::mat4 model_matrix( 1.0f ); // Identity matrix.
	
	// Define view matrix.
	// Transforms from world-space to camera-space.
	glm::mat4 view_matrix = glm::lookAt( camera.position,
										 camera.target,
										 camera.up );

	// Define projection matrix.
	// Transforms from camera-space to clip-space.
	glm::mat4 projection_matrix = glm::perspective( camera.fov_y,
													camera.resolution.x / camera.resolution.y,
													camera.near_clip,
													camera.far_clip );

	vertexShadeKernel<<< primitiveBlocks, tileSize >>>( device_vbo, vbosize,
														projection_matrix * view_matrix * model_matrix,
														camera.resolution,
														device_vbo_window_coords );
	hipDeviceSynchronize();

	//------------------------------
	// primitive assembly
	//------------------------------
	primitiveBlocks = ceil( ( ( float )ibosize / 3 ) / ( ( float )tileSize ) );
	primitiveAssemblyKernel<<< primitiveBlocks, tileSize >>>( device_vbo, vbosize,
															  device_cbo, cbosize,
															  device_ibo, ibosize,
															  device_nbo, nbosize,
															  device_vbo_window_coords,
															  primitives );
	hipDeviceSynchronize();

	//------------------------------
	// rasterization
	//------------------------------
	rasterizationKernel<<< primitiveBlocks, tileSize >>>( primitives,
														  ibosize / 3,
														  depthbuffer,
														  camera.resolution );
	hipDeviceSynchronize();

	//------------------------------
	// fragment shader
	//------------------------------
	fragmentShadeKernel<<< fullBlocksPerGrid, threadsPerBlock >>>( depthbuffer,
																   camera.resolution );
	hipDeviceSynchronize();

	//------------------------------
	// write fragments to framebuffer
	//------------------------------
	render<<< fullBlocksPerGrid, threadsPerBlock >>>( camera.resolution,
													  depthbuffer,
													  framebuffer );
	sendImageToPBO<<< fullBlocksPerGrid, threadsPerBlock >>>( PBOpos,
															  camera.resolution,
															  framebuffer );
	hipDeviceSynchronize();

	kernelCleanup();
	checkCUDAError("Kernel failed!");
}

void kernelCleanup(){
  hipFree( primitives );
  hipFree( device_vbo );
  hipFree( device_cbo );
  hipFree( device_ibo );
  hipFree( device_nbo );
  hipFree( framebuffer );
  hipFree( depthbuffer );
  hipFree( device_vbo_window_coords );
}