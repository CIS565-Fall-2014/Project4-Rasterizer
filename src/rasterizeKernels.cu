#include "hip/hip_runtime.h"
// CIS565 CUDA Rasterizer: A simple rasterization pipeline for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/random.h>
#include "rasterizeKernels.h"
#include "rasterizeTools.h"

glm::vec3* framebuffer;
fragment* depthbuffer;
float* device_vbo;
float* device_cbo;
int* device_ibo;
triangle* primitives;

cudaMat4 * projectionTransform;
cudaMat4 * MVtransform;
cudaMat4 * MVPtransform;


#define BLOCK_SIZE 16

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
   // exit(EXIT_FAILURE); 
  }
} 

//Handy dandy little hashing function that provides seeds for random number generation
__host__ __device__ unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Writes a given fragment to a fragment buffer at a given location
__host__ __device__ void writeToDepthbuffer(int x, int y, fragment frag, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    depthbuffer[index] = frag;
  }
}

//Reads a fragment from a given location in a fragment buffer
__host__ __device__ fragment getFromDepthbuffer(int x, int y, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return depthbuffer[index];
  }else{
    fragment f;
    return f;
  }
}

//Writes a given pixel to a pixel buffer at a given location
__host__ __device__ void writeToFramebuffer(int x, int y, glm::vec3 value, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    framebuffer[index] = value;
  }
}

//Reads a pixel from a pixel buffer at a given location
__host__ __device__ glm::vec3 getFromFramebuffer(int x, int y, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return framebuffer[index];
  }else{
    return glm::vec3(0,0,0);
  }
}

//Kernel that clears a given pixel buffer with a given color
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image, glm::vec3 color){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = color;
    }
}

//Kernel that clears a given fragment buffer with a given fragment
__global__ void clearDepthBuffer(glm::vec2 resolution, fragment* buffer, fragment frag){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      fragment f = frag;
      f.position.x = x;
      f.position.y = y;
      buffer[index] = f;
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;      
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//vertex shader
__global__ void vertexShadeKernel(float* vbo, int vbosize, cudaMat4 * MVP){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index<vbosize/3){

	  glm::vec4 oldP = glm::vec4(vbo[3*index],vbo[3*index + 1],vbo[3*index +2], 1.0f);
	  glm::vec4 newP = multiplyMV4(*MVP,oldP); 
	  vbo[3*index] = newP.x/newP.w;
	  vbo[3*index+1] = newP.y/newP.w;
	  vbo[3*index+2] = newP.z/newP.w;

  }
}

//Primative assembly
__global__ void primitiveAssemblyKernel(float* vbo, int vbosize, float* cbo, int cbosize, int* ibo, int ibosize, triangle* primitives){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int primitivesCount = ibosize/3;
  if(index<primitivesCount){

	  int i0,i1,i2;

	  i0 = 3 * ibo[3*index];
	  i1 = 3 * ibo[3*index + 1];
	  i2 = 3 * ibo[3*index + 2];

	  primitives[index].p0 = glm::vec3(vbo[i0],vbo[i0+1],vbo[i0+2]);
	  primitives[index].p1 = glm::vec3(vbo[i1],vbo[i1+1],vbo[i1+2]);
	  primitives[index].p2 = glm::vec3(vbo[i2],vbo[i2+1],vbo[i2+2]);

	  primitives[index].c0 = glm::vec3(cbo[i0],cbo[i0+1],cbo[i0+2]);
	  primitives[index].c1 = glm::vec3(cbo[i1],cbo[i1+1],cbo[i1+2]);
	  primitives[index].c2 = glm::vec3(cbo[i2],cbo[i2+1],cbo[i2+2]);
	 
  }
}

//TODO: Implement a rasterization method, such as scanline.
__global__ void rasterizationKernel(triangle* primitives, int primitivesCount, fragment* depthbuffer, glm::vec2 resolution){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(index<primitivesCount){
		
		int totalPixel = resolution.x * resolution.y;
		float halfResoX =  0.5f * (float) resolution.x;
		float halfResoY =  0.5f * (float) resolution.y;
		glm::vec3 normal = glm::normalize(glm::cross(primitives[index].p1 - primitives[index].p0, primitives[index].p2 - primitives[index].p1));

		if(normal.z >= 0.0f) return;

		glm::vec3 Min,Max;
		getAABBForTriangle(primitives[index],Min,Max);
		float pixelWidth = 1.0f/(float) resolution.x;
		float pixelHeight = 1.0f/(float) resolution.y;

		//loop thru all pixels in the bounding box
		for(int i = 0;i < (Max.x - Min.x)/pixelWidth + 1; i++)
		{
			for(int j = 0;j <(Max.y - Min.y)/pixelHeight + 1; j++)
			{
				glm::vec2 pixelPos = glm::vec2(Min.x + (float)i * pixelWidth, Min.y + (float)j * pixelHeight);
				glm::vec3 pixelBaryPos = calculateBarycentricCoordinate(primitives[index], pixelPos);
				
				//not in triangle
				if(!isBarycentricCoordInBounds(pixelBaryPos))
				{
					continue;
				}

				else
				{
					int x,y, pixelIndex;
					x = pixelPos.x * halfResoX + halfResoX;
					y = pixelPos.y  * halfResoY+ halfResoY;
					pixelIndex = x + y * resolution.x;

					float pixelDepth = getZAtCoordinate(pixelBaryPos, primitives[index]);

					if(pixelIndex < totalPixel && pixelIndex >= 0)// && pixelDepth < depthbuffer[pixelIndex].position.z)
					{
						depthbuffer[pixelIndex].position = glm::vec3(pixelPos.x,pixelPos.y,pixelDepth);
						depthbuffer[pixelIndex].normal = normal;
						//depthbuffer[pixelIndex].color = pixelBaryPos.x * primitives[index].c0 + pixelBaryPos.y * primitives[index].c1 + pixelBaryPos.z * primitives[index].c2;
						depthbuffer[pixelIndex].color = glm::vec3(0.0f,1.0f,0.0f);
					}

				}

			}
		}
		

		//Only display vertices
		/*
		  int x0,y0,x1,y1,x2,y2, P0,P1,P2;
		  x0 = primitives[index].p0.x * 0.5f * resolution.x + 0.5f * resolution.x;
		  x1 = primitives[index].p1.x * 0.5f * resolution.x + 0.5f * resolution.x;
		  x2 = primitives[index].p2.x * 0.5f * resolution.x + 0.5f * resolution.x;

		  y0 = primitives[index].p0.y * 0.5f * resolution.y + 0.5f * resolution.y;
		  y1 = primitives[index].p1.y * 0.5f * resolution.y + 0.5f * resolution.y;
		  y2 = primitives[index].p2.y * 0.5f * resolution.y + 0.5f * resolution.y;

		  //reversed y 
		  P0 = x0 + (y0) * resolution.x;
		  P1 = x1 + (y1) * resolution.x;
		  P2 = x2 + (y2) * resolution.x;

		  int totalPixel = resolution.x * resolution.y;
		  
		  if(P0 < totalPixel && P0 >=0) depthbuffer[P0].color = primitives[index].c0;
		  if(P1 < totalPixel && P1 >=0)depthbuffer[P1].color = primitives[index].c1;
		  if(P2 < totalPixel && P2 >=0)depthbuffer[P2].color = primitives[index].c2;
		*/
	}
}

//TODO: Implement a fragment shader
__global__ void fragmentShadeKernel(fragment* depthbuffer, light Light, glm::vec2 resolution){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  if(x<=resolution.x && y<=resolution.y){

	//  glm::vec3 L = glm::normalize(Light.position - depthbuffer[index].position);

	//  float coe = glm::dot(L,depthbuffer[index].normal);
	//  coe = (coe<0.0f) ? 0.0f:coe;
	 // coe = (coe>1.0f) ? 1.0f:coe;

	 // coe = abs(depthbuffer[index].position.z/2.0f);
	//  depthbuffer[index].color *= coe * Light.color;
  }
}

//Writes fragment colors to the framebuffer
__global__ void render(glm::vec2 resolution, fragment* depthbuffer, glm::vec3* framebuffer){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if(x<=resolution.x && y<=resolution.y){
    framebuffer[index] = depthbuffer[index].color;
  }
}

// Wrapper for the __global__ call that sets up the kernel calls and memory management
void cudaRasterizeCore(uchar4* PBOpos, glm::vec2 resolution, float frame, float* vbo, int vbosize, float* cbo, int cbosize, int* ibo, int ibosize, glm::mat4 glmProjectionTransform, glm::mat4 glmMVtransform,light Light){
	
	projectionTransform = new cudaMat4;
	MVtransform = new cudaMat4;
	MVPtransform = new cudaMat4;

	cudaMat4 * dev_projectionTransform;
	cudaMat4 * dev_MVtransform;
	cudaMat4 * dev_MVPtransform;
  
	*projectionTransform = utilityCore::glmMat4ToCudaMat4(glmProjectionTransform);
	*MVtransform = utilityCore::glmMat4ToCudaMat4(glmMVtransform);
	*MVPtransform =utilityCore::glmMat4ToCudaMat4(glmProjectionTransform * glmMVtransform);

	hipMalloc((void**) & dev_projectionTransform, sizeof(cudaMat4));
	hipMalloc((void**) & dev_MVtransform, sizeof(cudaMat4));
	hipMalloc((void**) & dev_MVPtransform, sizeof(cudaMat4));

	hipMemcpy(dev_projectionTransform,projectionTransform,sizeof(cudaMat4),hipMemcpyHostToDevice);
	hipMemcpy(dev_MVtransform,MVtransform,sizeof(cudaMat4),hipMemcpyHostToDevice);
	hipMemcpy(dev_MVPtransform,MVPtransform,sizeof(cudaMat4),hipMemcpyHostToDevice);

	light transformedLight = Light;
	transformedLight.position = multiplyMV(*MVPtransform,glm::vec4(Light.position,1.0f));

	// set up crucial magic
	int tileSize = 8;
	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(resolution.x)/float(tileSize)), (int)ceil(float(resolution.y)/float(tileSize)));

	//set up framebuffer
	framebuffer = NULL;
	hipMalloc((void**)&framebuffer, (int)resolution.x*(int)resolution.y*sizeof(glm::vec3));
  
	//set up depthbuffer
	depthbuffer = NULL;
	hipMalloc((void**)&depthbuffer, (int)resolution.x*(int)resolution.y*sizeof(fragment));

	//kernel launches to black out accumulated/unaccumlated pixel buffers and clear our scattering states
	clearImage<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, framebuffer, glm::vec3(0,0,0));
  
	fragment frag;
	frag.color = glm::vec3(0,0,0);
	frag.normal = glm::vec3(0,0,0);
	frag.position = glm::vec3(0,0,-10000);
	clearDepthBuffer<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer,frag);

	//------------------------------
	//memory stuff
	//------------------------------
	primitives = NULL;
	hipMalloc((void**)&primitives, (ibosize/3)*sizeof(triangle));

	device_ibo = NULL;
	hipMalloc((void**)&device_ibo, ibosize*sizeof(int));
	hipMemcpy( device_ibo, ibo, ibosize*sizeof(int), hipMemcpyHostToDevice);

	device_vbo = NULL;
	hipMalloc((void**)&device_vbo, vbosize*sizeof(float));
	hipMemcpy( device_vbo, vbo, vbosize*sizeof(float), hipMemcpyHostToDevice);

	device_cbo = NULL;
	hipMalloc((void**)&device_cbo, cbosize*sizeof(float));
	hipMemcpy( device_cbo, cbo, cbosize*sizeof(float), hipMemcpyHostToDevice);

	tileSize = 32;
	int primitiveBlocks = ceil(((float)vbosize/3)/((float)tileSize));

	//------------------------------
	//vertex shader
	//------------------------------
	vertexShadeKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize,dev_MVPtransform);

	hipDeviceSynchronize();
	//------------------------------
	//primitive assembly
	//------------------------------
	primitiveBlocks = ceil(((float)ibosize/3)/((float)tileSize));
	primitiveAssemblyKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize, device_cbo, cbosize, device_ibo, ibosize, primitives);

	hipDeviceSynchronize();
	//------------------------------
	//rasterization
	//------------------------------
	rasterizationKernel<<<primitiveBlocks, tileSize>>>(primitives, ibosize/3, depthbuffer, resolution);

	hipDeviceSynchronize();
	//------------------------------
	//fragment shader
	//------------------------------
	fragmentShadeKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(depthbuffer, transformedLight, resolution);

	hipDeviceSynchronize();
	//------------------------------
	//write fragments to framebuffer
	//------------------------------
	render<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer, framebuffer);
	sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, resolution, framebuffer);

	hipDeviceSynchronize();

	kernelCleanup();

	hipFree(dev_projectionTransform);
	hipFree(dev_MVtransform);
	hipFree(dev_MVPtransform);

	delete projectionTransform,MVtransform,MVPtransform;


	checkCUDAError("Kernel failed!");
}

void kernelCleanup(){
  hipFree( primitives );
  hipFree( device_vbo );
  hipFree( device_cbo );
  hipFree( device_ibo );
  hipFree( framebuffer );
  hipFree( depthbuffer );

}

